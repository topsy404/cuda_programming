#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include <iostream>

using std::cout;

__global__ void vectorAdd(int *a, int *b, int *c, int N){
    // Calculate global thread ID
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    // Boundary check
    if(tid < N){
        c[tid] = a[tid] + b[tid];
    }
}

int main(){
    // Array size of 2^16
    const int N = 1 << 16;
    size_t bytes = N * sizeof(int);

    // Declare unified memory pointers
    int *a, *b, *c;

    // Allocate memory for these pointers
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    // Get the device ID for prefetching calls
    int id = hipGetDevice(&id);

    // Set some hints about the data and do some prefetching 
    hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, id);

    // Initialze vectors
    for(int i = 0; i < N; i++){
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    // Pre-fetch 'a' and 'b' array to the specified deivce (GPU)
    hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
    hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
    hipMemPrefetchAsync(a, bytes, id);
    hipMemPrefetchAsync(b, bytes, id);

    // Threads per CTA 
    int BLOCK_SIZE = 1 << 10;
    int GRID_SIZE = (N + BLOCK_SIZE - 1)/BLOCK_SIZE;

    vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);

    // Wait for all previous operations before using values
    hipDeviceSynchronize();

    // Prefetch to the host (CPU)
    hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

    // Verify the result on CPU
    for(int i = 0; i < N; i++){
        if(c[i] != a[i] + b[i]){
            cout << "c is " << c[i] << " ; a + b is : " << a[i] + b[i] << std::endl;
        }
    }

    // Free unified memory
    hipFree(a);
    hipFree(b);
    hipFree(c);

    cout << "Completed\n";

    return 0;

}
